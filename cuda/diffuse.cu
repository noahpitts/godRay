#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"
#include "commonStructs.h"

using namespace optix;

rtDeclareVariable( float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable( float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable( float4, geometry_color, attribute geometry_color, );

rtDeclareVariable(optix::Ray, ray,   rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );

rtDeclareVariable( float3, Kd, , );
rtDeclareVariable(rtObject,      top_object, , );

rtBuffer<DirectionalLight> light_buffer;

RT_PROGRAM void any_hit_shadow()
{
    prd_shadow.attenuation = make_float3( 0.0f );
    rtTerminateRay();
}

// Note: both the hemisphere and direct light sampling below use pure random numbers to avoid any patent issues.
// Stratified sampling or QMC would improve convergence.  Please keep this in mind when judging noise levels.

RT_PROGRAM void closest_hit_radiance()
{

    const float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    const float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    const float z1 = rnd( prd_radiance.seed );
    const float z2 = rnd( prd_radiance.seed );
    
    float3 w_in;
    optix::cosine_sample_hemisphere( z1, z2, w_in );
    const optix::Onb onb( ffnormal );
    onb.inverse_transform( w_in );
    const float3 fhp = rtTransformPoint( RT_OBJECT_TO_WORLD, front_hit_point );

    prd_radiance.origin = front_hit_point;
    prd_radiance.direction = w_in;
    
    prd_radiance.attenuation *= Kd * make_float3( geometry_color );

    // Add direct light sample weighted by shadow term and 1/probability.
    // The pdf for a directional area light is 1/solid_angle.

    const DirectionalLight& light = light_buffer[0];
    const float3 light_center = fhp + light.direction;
    const float r1 = rnd( prd_radiance.seed );
    const float r2 = rnd( prd_radiance.seed );
    const float2 disk_sample = square_to_disk( make_float2( r1, r2 ) );
    const float3 jittered_pos = light_center + light.radius*disk_sample.x*light.v0 + light.radius*disk_sample.y*light.v1;
    const float3 L = normalize( jittered_pos - fhp );

    const float NdotL = dot( ffnormal, L);
    if(NdotL > 0.0f) {
        PerRayData_shadow shadow_prd;
        shadow_prd.attenuation = make_float3( 1.0f );
        optix::Ray shadow_ray ( fhp, L, /*shadow ray type*/ 1, 0.0f );
        rtTrace(top_object, shadow_ray, shadow_prd);

        const float solid_angle = light.radius*light.radius*M_PIf;
        prd_radiance.radiance += NdotL * light.color * solid_angle * shadow_prd.attenuation;
    }
    

}

