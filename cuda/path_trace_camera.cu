#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"

using namespace optix;


rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(float3,        cutoff_color, , );
rtDeclareVariable(int,           max_depth, , );
rtBuffer<uchar4, 2>              output_buffer;
rtBuffer<float4, 2>              accum_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  frame, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );


__inline__ __device__ float3 tonemap( const float3 in )
{
    // hard coded exposure for sun/sky
    const float exposure = 1.0f/30.0f;
    float3 x = exposure*in;

    // "filmic" map from a GDC talk by John Hable.  This includes 1/gamma.
    x = fmaxf( x - make_float3(0.004f), make_float3(0.0f) ); 
    float3 ret = (x*(6.2f*x + make_float3(.5f)))/(x*(6.2f*x + make_float3(1.7f)) + make_float3(0.06f));

    return ret;
}

RT_PROGRAM void pinhole_camera()
{

  size_t2 screen = output_buffer.size();
  unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame);

  // Subpixel jitter: send the ray through a different position inside the pixel each time,
  // to provide antialiasing.
  float2 subpixel_jitter = frame == 0 ? make_float2( 0.0f ) : make_float2(rnd( seed ) - 0.5f, rnd( seed ) - 0.5f);

  float2 d = (make_float2(launch_index) + subpixel_jitter) / make_float2(screen) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);

  PerRayData_radiance prd;
  prd.depth = 0;
  prd.seed = seed;
  prd.done = false;

  // These represent the current shading state and will be set by the closest-hit or miss program

  // brdf attenuation from surface interaction
  prd.attenuation = make_float3( 1.0f );

  // light from a light source or miss program
  prd.radiance = make_float3( 0.0f );

  // next ray to be traced
  prd.origin = make_float3( 0.0f );
  prd.direction = make_float3( 0.0f );

  float3 result = make_float3( 0.0f );

  // Main render loop. This is not recursive, and for high ray depths
  // will generally perform better than tracing radiance rays recursively
  // in closest hit programs.
  for(;;) {
      optix::Ray ray(ray_origin, ray_direction, /*ray type*/ 0, scene_epsilon );
      rtTrace(top_object, ray, prd);

      result += prd.attenuation * prd.radiance;

      if ( prd.done ) {
          break;
      } else if ( prd.depth >= max_depth ) {
        result += prd.attenuation * cutoff_color;
        break;
      }

      prd.depth++;

      // Update ray data for the next path segment
      ray_origin = prd.origin;
      ray_direction = prd.direction;
  }

  float4 acc_val = accum_buffer[launch_index];
  if( frame > 0 ) {
    acc_val = lerp( acc_val, make_float4( result, 0.f ), 1.0f / static_cast<float>( frame+1 ) );
  } else {
    acc_val = make_float4( result, 0.f );
  }
  output_buffer[launch_index] = make_color( tonemap( make_float3( acc_val ) ) );
  accum_buffer[launch_index] = acc_val;
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  output_buffer[launch_index] = make_color( bad_color );
}

