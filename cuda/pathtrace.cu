#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../inc/helpers.h"
#include "../inc/random.h"
#include "../inc/prd.h"

using namespace optix;

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(int, max_depth, , );
rtBuffer<uchar4, 2> output_buffer;
rtBuffer<float4, 2> accum_buffer;
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

// ------------------------
// PIXEL SAMPLING FUNCTIONS
// ------------------------

// Subpixel jitter: send the ray through a different position inside the pixel each time
__inline__ __device__ float2 samplePixel_jitter(unsigned int *seed)
{
  size_t2 screen = output_buffer.size();
  *seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame);
  float2 subpixel_jitter = frame == 0 ? make_float2(0.0f) : make_float2(rnd(*seed) - 0.5f, rnd(*seed) - 0.5f);

  return (make_float2(launch_index) + subpixel_jitter) / make_float2(screen) * 2.f - 1.f;
}

// -----------------------------
// CAMERA RAY FUNCTIONS
// -----------------------------

// Pinhole Camera
__inline__ __device__ float genPinholeCameraRay(float3* o, float3* d, float2 sample_pt)
{
  *o = eye;
  *d = normalize(sample_pt.x * U + sample_pt.y * V + W);
  return 1.0f;
}

// ThinLens Camera - TODO

// ---------------------------
// INTEGRATOR FUNCTIONS
// ---------------------------

// Iterative Pathtracer
__inline__ __device__ float3 Li_pathtrace(float3 ray_origin, float3 ray_dir, unsigned int seed)
{
  // Initialize per ray data structure
  PerRayData_radiance prd;
  prd.depth = 0;
  prd.seed = seed;
  //prd.done = false;

  int min_depth = 3;

  //prd.in_media = 0;
  prd.beta = make_float3(1.0f);
  prd.radiance = make_float3(0.0f); // light from a light source or miss program

  // next ray to be traced
  prd.origin = make_float3(0.0f);
  prd.direction = make_float3(0.0f);

  float3 L = make_float3(0.0f);

  // pathtrace loop. 
  for (;;)
  {
    // intersect ray with scene and store intersection radiance and attenuation(beta)
    optix::Ray ray(ray_origin, ray_dir, /*ray type*/ 0, scene_epsilon);
    rtTrace(top_object, ray, prd);

    L += prd.beta * prd.radiance;

    // terminate path if no more contribution
    //if (prd.beta.x <= 0.001f && prd.beta.y <= 0.001f && prd.beta.z <= 0.001f) prd.done = true;
    //if (prd.done)
    //{
    //  break;
    //}
    // terminate path if max depth was reached
    if (prd.depth >= max_depth)
    {
      //L += prd.beta * cutoff_color;
      break;
    }
    // russian roulette termination | pbrt 879
    if (prd.depth > min_depth)
    {
      float q = 1.0f - prd.beta.y;
      if (q < 0.05f) q = 0.05f;
      if (rnd(prd.seed) < q)
        break;
      prd.beta /= 1.0f - q;
    }

    prd.depth++;

    // Update ray data for the next path segment
    ray_origin = prd.origin;
    ray_dir = prd.direction;
  }

  return L;
}

// ----------------------
// TONE MAPPING FUNCTIONS
// ----------------------
__inline__ __device__ float3 tonemap(const float3 in)
{
  // hard coded exposure for sun/sky
  const float exposure = 1.0f / 30.0f;
  float3 x = exposure * in;

  // "filmic" map from a GDC talk by John Hable.  This includes 1/gamma.
  x = fmaxf(x - make_float3(0.004f), make_float3(0.0f));
  float3 ret = (x * (6.2f * x + make_float3(.5f))) / (x * (6.2f * x + make_float3(1.7f)) + make_float3(0.06f));

  return ret;
}

RT_PROGRAM void render_pixel()
{
  // seed for random num generator
  unsigned int seed;

  // Initialize camera sample for current sample | pbrt 30
  float2 cameraSample = samplePixel_jitter(&seed);

  // Generate Camera Ray for current Sample | pbrt 31
  float3 ray_origin; 
  float3 ray_dir;
  float ray_weight = genPinholeCameraRay(&ray_origin, &ray_dir, cameraSample); // ray_weight is used for vignetting

  // Evaluate Radiance along Camera Ray | pbrt 31
  float3 L = make_float3(0.0f);
  if (ray_weight > 0.0f) 
    L = Li_pathtrace(ray_origin, ray_dir, seed) * ray_weight;


  // ACCUMULATE AND OUTPUT TO IMAGE

  float4 acc_val = accum_buffer[launch_index];
  if (frame > 0)
  {
    acc_val = lerp(acc_val, make_float4(L, 0.f), 1.0f / static_cast<float>(frame + 1));
  }
  else
  {
    acc_val = make_float4(L, 0.f);
  }
  output_buffer[launch_index] = make_color( tonemap( make_float3( acc_val ) ) );
  // output_buffer[launch_index] = make_color(make_float3(acc_val));
  accum_buffer[launch_index] = acc_val;
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  switch(code) {
    case RT_EXCEPTION_TEXTURE_ID_INVALID: rtPrintf("TEXTURE_ID\n"); break;
    case RT_EXCEPTION_BUFFER_ID_INVALID: rtPrintf("BUFFER_ID\n"); break;
    case RT_EXCEPTION_INDEX_OUT_OF_BOUNDS: rtPrintf("INDEX_OUT_OF_BOUNDS\n"); break;
    case RT_EXCEPTION_STACK_OVERFLOW: rtPrintf("STACK_OVERFLOW\n"); break;
    case RT_EXCEPTION_BUFFER_INDEX_OUT_OF_BOUNDS: rtPrintf("BUFFER_INDEX_OUT_OF_BOUNDS\n"); break;
    case RT_EXCEPTION_INVALID_RAY: rtPrintf("INVALID_RAY\n"); break;
    case RT_EXCEPTION_INTERNAL_ERROR: rtPrintf("INTERNAL_ERROR\n"); break;
    case RT_EXCEPTION_USER: rtPrintf("USER\n"); break;
    case RT_EXCEPTION_ALL: rtPrintf("ALL\n"); break;
    default: rtPrintf("UNKNOWN Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y);
  }
  output_buffer[launch_index] = make_color(bad_color);
}


