#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


// Intersection and bounds programs for custom box prims as the leaf nodes of a
// BVH.  The boxes are represented as 4 bytes per box (VOX format).

// Note: 
// This is a compromise between intersection cost and memory cost: a BVH
// built over triangles could be faster to intersect, but would use more memory
// for the leaves (12 triangles per box), while an intersector that handled
// larger chunks of boxes via 3d grid traversal would be slower but would
// reduce the size of the BVH.  The compromise solution seems fine for a single
// VOX file (max 256**3 boxes) or a handful of files.  The custom 3d grid
// intersector could be interesting for very large scenes, e.g., Minecraft
// files, that might have far more than 256**3 boxes.


#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

#include "intersection_refinement.h"

using namespace optix;

// 8-bit indices as in VOX format.  We expand these into floating point coords during intersection.
rtBuffer< optix::uchar4 > box_buffer;

rtBuffer< optix::uchar4 > palette_buffer;

rtDeclareVariable( float3, anchor, , ) = {0.0f, 0.0f, 0.0f};

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable( float3, back_hit_point, attribute back_hit_point, );
rtDeclareVariable( float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable( float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable( float4, geometry_color, attribute geometry_color, ); 

static __device__ float3 boxnormal(float3 boxmin, float3 boxmax, float t)
{
    float3 t0 = (boxmin - ray.origin)/ray.direction;
    float3 t1 = (boxmax - ray.origin)/ray.direction;
    float3 neg = make_float3(t==t0.x?1:0, t==t0.y?1:0, t==t0.z?1:0);
    float3 pos = make_float3(t==t1.x?1:0, t==t1.y?1:0, t==t1.z?1:0);
    return pos-neg;
}

// Note: might be more efficient to combine with intersection
static __device__ float3 boxanchor(float3 boxmin, float3 boxmax, float t)
{
    float3 t0 = (boxmin - ray.origin) / ray.direction;
    if ( t == t0.x || t == t0.y || t == t0.z ) return boxmin;
    return boxmax;
}

static __device__ __inline__ float4 make_float4( uchar4 c )
{
    return make_float4( c.x, c.y, c.z, c.w );
}

RT_PROGRAM void intersect( int primId )
{
    // Expand cell in unit box
    const uchar4 b = box_buffer[primId];
    const float3 inv_box_dims = make_float3( 1.0f / 255.0f );
    const float3 boxmin = anchor + make_float3( b.x, b.y, b.z ) * inv_box_dims;
    const float3 boxmax = boxmin + inv_box_dims;

    float3 t0 = (boxmin - ray.origin)/ray.direction;
    float3 t1 = (boxmax - ray.origin)/ray.direction;
    float3 near = fminf(t0, t1);
    float3 far = fmaxf(t0, t1);
    float tmin = fmaxf( near );
    float tmax = fminf( far );

    if(tmin <= tmax) {
        bool check_second = true;
        if( rtPotentialIntersection( tmin ) ) {
            unsigned char color_index = box_buffer[primId].w;
            geometry_color = make_float4( palette_buffer[ color_index ] ) * ( 1.0f / 255.0f );
            shading_normal = geometric_normal = boxnormal( boxmin, boxmax, tmin );

            const float3 anchor = boxanchor( boxmin, boxmax, tmin );
            refine_and_offset_hitpoint( ray.origin + tmin*ray.direction, ray.direction,
                    shading_normal, anchor,
                    back_hit_point, front_hit_point );

            if(rtReportIntersection(0))
                check_second = false;
        } 
        if(check_second) {
            if( rtPotentialIntersection( tmax ) ) {
                unsigned char color_index = box_buffer[primId].w;
                geometry_color = make_float4( palette_buffer[ color_index ] ) * ( 1.0f / 255.0f );
                shading_normal = geometric_normal = boxnormal( boxmin, boxmax, tmax );

                const float3 anchor = boxanchor( boxmin, boxmax, tmax );
                refine_and_offset_hitpoint( ray.origin + tmax*ray.direction, ray.direction,
                        shading_normal, anchor,
                        back_hit_point, front_hit_point );

                rtReportIntersection(0);
            }
        }
    }
}

RT_PROGRAM void bounds (int primId, float result[6])
{
    const uchar4 b = box_buffer[primId];
    const float3 inv_box_dims = make_float3( 1.0f / 255.0f );
    const float3 boxmin = anchor + make_float3( b.x, b.y, b.z ) * inv_box_dims;
    const float3 boxmax = boxmin + inv_box_dims;

    optix::Aabb* aabb = (optix::Aabb*)result;
    aabb->set( boxmin, boxmax );
}

